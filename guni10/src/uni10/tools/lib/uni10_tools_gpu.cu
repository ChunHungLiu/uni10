#include "hip/hip_runtime.h"
#include <uni10/tools/uni10_tools.h>
#include <stdio.h>
#include <string.h>

namespace uni10{
size_t MEM_USAGE = 0;
size_t GPU_MEM_USAGE = 0;
const size_t GPU_MEM_MAX = GPU_GLOBAL_MEM / 2;
void* elemAlloc(size_t memsize, bool& ongpu){
	void* ptr = NULL;
	if(GPU_MEM_USAGE + memsize <= GPU_MEM_MAX){
		hipError_t cuflag = hipMalloc(&ptr, memsize);
		assert(cuflag == hipSuccess);
		//printf("GPU: %d used, allocate %d bytes, %x\n", GPU_MEM_USAGE, memsize, ptr);
		GPU_MEM_USAGE += memsize;
		ongpu = true;
	}else{
		//printf("CPU: %d used, allocate %d bytes, %x\n", MEM_USAGE, memsize, ptr);
		ptr = malloc(memsize);
		assert(ptr != NULL);
		MEM_USAGE += memsize;
		ongpu = false;
	}
	return ptr;
}

void* elemAllocForce(size_t memsize, bool ongpu){
	void* ptr = NULL;
	if(ongpu){
		hipError_t cuflag = hipMalloc(&ptr, memsize);
		assert(cuflag == hipSuccess);
		GPU_MEM_USAGE += memsize;
	}
	else{
		ptr = malloc(memsize);
		assert(ptr != NULL);
		MEM_USAGE += memsize;
	}
	return ptr;
}

void* elemCopy(void* des, const void* src, size_t memsize, bool des_ongpu, bool src_ongpu){
	hipError_t cuflag;
	if((des_ongpu)){
		if(src_ongpu){
			cuflag = hipMemcpy(des, src, memsize, hipMemcpyDeviceToDevice);
			//printf("memcpy %x to %x D2D\n", src, des);
		}
		else{
			cuflag = hipMemcpy(des, src, memsize, hipMemcpyHostToDevice);
			//printf("memcpy %x to %x H2D\n", src, des);
		}
		assert(cuflag == hipSuccess);
	}else{
		if(src_ongpu){
			cuflag = hipMemcpy(des, src, memsize, hipMemcpyDeviceToHost);
			//printf("memcpy %x to %x D2H\n", src, des);
			assert(cuflag == hipSuccess);
		}
		else{
			memcpy(des, src, memsize);
			//printf("memcpy H2H\n");
		}
	}
	return des;
}

void elemFree(void* ptr, size_t memsize, bool ongpu){
	hipError_t cuflag;
	assert(ptr != NULL);
	if(ongpu){
		//printf("FREE(%x) %d from GPU, %d used\n", ptr, memsize, GPU_MEM_USAGE);
		cuflag = hipFree(ptr);
		assert(cuflag == hipSuccess);
		GPU_MEM_USAGE -= memsize;
	}else{
		//printf("FREE %d from CPU, %d used\n", memsize, MEM_USAGE);
		free(ptr);
		MEM_USAGE -= memsize;
	}
	ptr = NULL;
}
void elemBzero(void* ptr, size_t memsize, bool ongpu){
	if(ongpu)
		hipMemset(ptr, 0, memsize);
	else
		memset(ptr, 0, memsize);
}

__global__ void gpu_rand(double* elem, size_t N){
	size_t idx = blockIdx.y * BLOCKMAX * THREADMAX +  blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int r = (1664525 * ((1664525 * idx + 1013904223) % UINT_MAX) + 1013904223) % UINT_MAX;
	if(idx < N)
		elem[idx] = double(r) / UINT_MAX;
}

void elemRand(double* elem, size_t N, bool ongpu){
	if(ongpu){
		size_t blockNum = (N + THREADMAX - 1) / THREADMAX;
		dim3 gridSize(blockNum % BLOCKMAX, (blockNum + BLOCKMAX - 1) / BLOCKMAX);
		gpu_rand<<<gridSize, THREADMAX>>>(elem, N);
	}
	else{
		for(size_t i = 0; i < N; i++)
			elem[i] = ((double)rand()) / RAND_MAX; //lapack_uni01_sampler();
	}
}

__global__ void _setDiag(double* elem, double* diag_elem, size_t M, size_t N, size_t diag_N){
	size_t idx = blockIdx.y * BLOCKMAX * THREADMAX +  blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < diag_N && idx < M && idx < N)
		elem[idx * N + idx] = diag_elem[idx];
}

__global__ void _getDiag(double* elem, double* diag_elem, size_t M, size_t N, size_t diag_N){
	size_t idx = blockIdx.y * BLOCKMAX * THREADMAX +  blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < diag_N && idx < M && idx < N)
		diag_elem[idx] = elem[idx * N + idx];
}

void setDiag(double* elem, double* diag_elem, size_t M, size_t N, size_t diag_N, bool ongpu, bool diag_ongpu){
	if((ongpu)){
		size_t blockNum = (N + THREADMAX - 1) / THREADMAX;
		dim3 gridSize(blockNum % BLOCKMAX, (blockNum + BLOCKMAX - 1) / BLOCKMAX);
		if(diag_ongpu){
			_setDiag<<<gridSize, THREADMAX>>>(elem, diag_elem, M, N, diag_N);
		}
		else{
			size_t memsize = diag_N * sizeof(double);
			double* src_elem;
			hipError_t cuflag = hipMalloc(&src_elem, memsize);
			assert(cuflag == hipSuccess);
			cuflag = hipMemcpy(src_elem, diag_elem, memsize, hipMemcpyHostToDevice);
			assert(cuflag == hipSuccess);
			_setDiag<<<gridSize, THREADMAX>>>(elem, src_elem, M, N, diag_N);
			hipFree(src_elem);
		}
	}else{
		double* src_elem;
		if(diag_ongpu){
			size_t memsize = diag_N * sizeof(double);
			src_elem = (double*) malloc(memsize);
			hipError_t cuflag = hipMemcpy(src_elem, diag_elem, memsize, hipMemcpyDeviceToHost);
			assert(cuflag == hipSuccess);
		}
		else
			src_elem = diag_elem;
		int min = M < N ? M : N;
		min = min < diag_N ? min : diag_N;
		for(size_t i = 0; i < min; i++)
			elem[i * N + i] = src_elem[i];
	}
}

void getDiag(double* elem, double* diag_elem, size_t M, size_t N, size_t diag_N, bool ongpu, bool diag_ongpu){
	if((ongpu)){
		size_t blockNum = (N + THREADMAX - 1) / THREADMAX;
		dim3 gridSize(blockNum % BLOCKMAX, (blockNum + BLOCKMAX - 1) / BLOCKMAX);
		if(diag_ongpu){
			_getDiag<<<gridSize, THREADMAX>>>(elem, diag_elem, M, N, diag_N);
		}
		else{
			size_t memsize = diag_N * sizeof(double);
			double* tmp_elem;
			hipError_t cuflag = hipMalloc(&tmp_elem, memsize);
			assert(cuflag == hipSuccess);
			_getDiag<<<gridSize, THREADMAX>>>(elem, tmp_elem, M, N, diag_N);
			cuflag = hipMemcpy(diag_elem, tmp_elem, memsize, hipMemcpyHostToDevice);
			assert(cuflag == hipSuccess);
			hipFree(tmp_elem);
		}
	}else{
		double* tmp_elem;
		size_t memsize = diag_N * sizeof(double);
		if(diag_ongpu)
			tmp_elem = (double*)malloc(memsize);
		else
			tmp_elem = diag_elem;
		int min = M < N ? M : N;
		min = min < diag_N ? min : diag_N;
		for(size_t i = 0; i < min; i++)
			tmp_elem[i] = elem[i * N + i];
		if(diag_ongpu){
			hipError_t cuflag = hipMemcpy(diag_elem, tmp_elem, memsize, hipMemcpyDeviceToHost);
			assert(cuflag == hipSuccess);
		}
	}
}

void* mvGPU(void* elem, size_t memsize, bool& ongpu){
	if(!ongpu)
		if(GPU_MEM_USAGE + memsize <= GPU_MEM_MAX){
			void* newElem = elemAlloc(memsize, ongpu);
			elemCopy(newElem, elem, memsize, ongpu, false);
			elemFree(elem, memsize, false);
			elem = newElem;
		}
	return elem;
}

void* mvCPU(void* elem, size_t memsize, bool& ongpu){
	if(ongpu){
		double *newElem = (double*)malloc(memsize);
		elemCopy(newElem, elem, memsize, false, true);
		elemFree(elem, memsize, true);
		MEM_USAGE += memsize;
		ongpu = false;
		elem = newElem;
	}
	return elem;
}

void syncMem(void** elemA, void** elemB, size_t memsizeA, size_t memsizeB, bool& ongpuA, bool& ongpuB){	
	if((!ongpuA) || (!ongpuB)){
		size_t memsize = 0;
		if(!ongpuA)
			memsize += memsizeA;
		if(!ongpuB)
			memsize += memsizeB;
		if(GPU_MEM_USAGE + memsize <= GPU_MEM_MAX){
			if(!ongpuA)
				*elemA = mvGPU(*elemA, memsizeA, ongpuA);
			if(!ongpuB)
				*elemB = mvGPU(*elemB, memsizeB, ongpuB);
		}
		else{
			if(ongpuA)
				*elemA = mvCPU(*elemA, memsizeA, ongpuA);
			if(ongpuB)
				*elemB = mvCPU(*elemB, memsizeB, ongpuB);
		}
	}
}
void shrinkWithoutFree(size_t memsize, bool ongpu){
	printf("SHRINKING!!\n");
	if(ongpu)
		GPU_MEM_USAGE -= memsize;
	else
		MEM_USAGE -= memsize;
}

__global__ void _reshapeElem(double* oldElem, int bondNum, size_t elemNum, size_t* offset, double* newElem){
	size_t oldIdx = blockIdx.y * BLOCKMAX * THREADMAX +  blockIdx.x * blockDim.x + threadIdx.x;
	size_t idx = oldIdx;
	size_t newIdx = 0;
	if(idx < elemNum){
		for(int i = 0; i < bondNum; i++){
			newIdx += (idx/offset[i]) * offset[bondNum + i];
			idx = idx % offset[i];
		}
		newElem[newIdx] = oldElem[oldIdx];
	}
}
void reshapeElem(double* oldElem, int bondNum, size_t elemNum, size_t* offset, double* newElem){
	size_t* D_offset;
	assert(hipMalloc((void**)&D_offset, 2 * sizeof(size_t) * bondNum) == hipSuccess);
	assert(hipMemcpy(D_offset, offset, 2 * sizeof(size_t) * bondNum, hipMemcpyHostToDevice) == hipSuccess);
	size_t blockNum = (elemNum + THREADMAX - 1) / THREADMAX;
	dim3 gridSize(blockNum % BLOCKMAX, (blockNum + BLOCKMAX - 1) / BLOCKMAX);
	_reshapeElem<<<gridSize, THREADMAX>>>(oldElem, bondNum, elemNum, D_offset, newElem);
}


double getElemAt(size_t idx, double* elem, bool ongpu){
	if(ongpu){
		printf("YOHA get!!\n");
		double val;
		assert(hipMemcpy(&val, &(elem[idx]), sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
		return val;
	}	
	else
		return elem[idx];
}

void setElemAt(size_t idx, double val, double* elem, bool ongpu){
	if(ongpu){
		printf("YOHA set!!\n");
		assert(hipMemcpy(&(elem[idx]), &val, sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
	}
	else
		elem[idx] = val;
}

};	/* namespace uni10 */
