#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <uni10/tensor-network/Matrix.h>

// Device code
namespace uni10{
__global__ void VecAdd(double* A, double* B, double* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void vecAdd(double* h_A, double* h_B, int N, double* h_C){
    size_t size = N * sizeof(double);

    // Allocate vectors in device memory
    double* d_A;
    assert(hipMalloc(&d_A, size) == hipSuccess);
    double* d_B;
    hipMalloc(&d_B, size);
    double* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
            
    // Free host memory
}
}
